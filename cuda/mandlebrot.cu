#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_WIDTH 640
#define MAX_HEIGHT 480
#define SIZE 307200 //640*480
#define MAX_ITER 256
#define BLOCKS 4

__global__ void  processMandelbrotAsVector(int *work) {
	int iter, min, max;
	double x, y, zx, zy, zx2, zy2;
	min = MAX_ITER; max = 0;
	double scale = 1. / MAX_ITER;
	double cx = -.6, cy = 0;

	int start = (threadIdx.x * MAX_WIDTH) + 1;
	//int end = start + (int) floor(MAX_HEIGHT * MAX_WIDTH/THREADS_PER_BLOCKS);

	for (int i = start, j = 0; j < MAX_WIDTH; i++, j++) {
		y = (threadIdx.x - MAX_HEIGHT / 2) * scale + cy;
		x = (j - MAX_WIDTH / 2) * scale + cx;
		iter = 0;

		zx = hypot(x - .25, y);
		if (x < zx - 2 * zx * zx + .25) iter = MAX_ITER;
		if ((x + 1)*(x + 1) + y * y < 1 / 16) iter = MAX_ITER;

		zx = zy = zx2 = zy2 = 0;
		for (; iter < MAX_ITER && zx2 + zy2 < 4; iter++) {
			zy = 2 * zx * zy + y;
			zx = zx2 - zy2 + x;
			zx2 = zx * zx;
			zy2 = zy * zy;
		}
		if (iter < min) min = iter;
		if (iter > max) max = iter;
		work[i] = iter;
	}
}

void printVector(int *work) {
	for (int i = 0; i < SIZE; i++) {
		printf("[%i]: %i\n", i, work[i]);
	}
}


int main()
{
	
	hipError_t cudaStatus;
	int *work = 0;
	int *hwork = (int *) malloc(SIZE * sizeof(int));	
		
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto End;
	}

	cudaStatus = hipMalloc((void**)&work, sizeof(int) * SIZE);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto End;
	}

	processMandelbrotAsVector << <1, MAX_HEIGHT >> >(work);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto End;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto End;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hwork, work, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto End;
	}
	
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		goto End;
	}

	printVector(hwork);

End:
	char ch;
	printf("Programa finalizado. Digite qualquer tecla para continuar ... ");
	scanf("%c", &ch);

	return 0;


}
